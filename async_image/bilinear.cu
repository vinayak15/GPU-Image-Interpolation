#include "hip/hip_runtime.h"
//Asynchronous data transfer for further optimization in bilinear interpolation  
//here code is same as compared to shared memory optimization except for few changes

#include<Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include<device_atomic_functions.h>
#include<hip/device_functions.h>
#include<stdlib.h>


//here we are using flag to determine which part of image is to be interpolated

__global__ void bilinear1(unsigned char *new_image, const unsigned char *image, int rows, int cols,int flag)  
{
	int index;
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;
	 
	index = (row + rows*flag / 4)*cols / 2 + col;
	row += flag*rows / 4;
	row *= 2;
	col *= 2;

	unsigned char a = image[index];
	new_image[(row)*cols + col] = a;
	__syncthreads();

}
__global__ void bilinear2(unsigned char *new_image, const unsigned char *image, int rows, int cols,int flag)
{
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;


	row += flag*rows / 4;
	row *= 2;
	col *= 2;

	new_image[(row + 1)*cols + col + 1] = (new_image[(row + 2)*cols + col] + new_image[(row*cols + col + 2)] + new_image[(row + 2)*cols + col + 2] + new_image[(row*cols + col)]) / 4;

	__syncthreads();

}

__global__ void bilinear3(unsigned char *new_image, const unsigned char *image, int rows, int cols,int flag)
{
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;
	row += flag*rows / 4;
	row *= 2;
	col *= 2;

	new_image[row*cols + col + 1] = (new_image[row*cols + col] + new_image[(row - 1)*cols + col + 1] + new_image[row*cols + col + 2] + new_image[(row + 1)*cols + col + 1]) / 4;

	new_image[(row + 1)*cols + col] = (new_image[row*cols + col] + new_image[(row + 1)*cols + col - 1] + new_image[(row + 1)*cols + col + 1] + new_image[(row + 2)*cols + col]) / 4;
	__syncthreads();

}

#include "hip/hip_runtime.h"
//Nearest Neighbour interpolation technique implemented on an image with shared memory 
//In this interpolation we are taking the image and dividing it in two parts upper half and lower half  
//Main reason for dividing is we cant have full image in our shared memeory array of pixels.
#include<Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include<device_atomic_functions.h>
#include<hip/device_functions.h>

__global__ void lneigh(unsigned char *new_image, const unsigned char *image, int rows, int cols,int flag)
{
	__shared__ unsigned char a[256 * 128];								//here half part of image is taken and interpolated because of memeory concerns of our GPU Hardware
	int index;
	int	col = threadIdx.x + blockDim.x*blockIdx.x; 				
	int row=threadIdx.y + blockDim.y*blockIdx.y;

	index = (row + rows*flag / 4)*cols/2 + col;						//index of image is decided to be intrpolated
	int index1 = row*cols/2 + col;
	row += flag*rows / 4;
	row *= 2;
	col *= 2;

	a[index1] = image[index];
	new_image[(row)*cols + col] = a[index1];						//interpolation happens
	new_image[(row)*cols + col + 1] = a[index1];
	new_image[(row + 1 )*cols + col] = a[index1];
	new_image[(row + 1)*cols + col + 1] = a[index1];
}

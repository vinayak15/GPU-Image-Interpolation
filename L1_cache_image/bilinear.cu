#include "hip/hip_runtime.h"
//Bilinear image interpolation using L1 cache optimization
//Here there is not much difference in code as compared to register memory optimization


#include<Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include<device_atomic_functions.h>
#include<hip/device_functions.h>
#include<stdlib.h>
__global__ void bilinear1(unsigned char *new_image, const unsigned char *image, int rows, int cols)
{
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;
	int index = row*cols / 2 + col;

	row *= 2;
	col *= 2;
	new_image[row*cols + col] = image[index];
	__syncthreads();

}
__global__ void bilinear2(unsigned char *new_image, const unsigned char *image, int rows, int cols)
{
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;
//	int index = row*cols / 2 + col;


	row *= 2;
	col *= 2;

	new_image[(row + 1)*cols + col + 1] = (new_image[(row + 2)*cols + col] + new_image[(row*cols + col + 2)] + new_image[(row + 2)*cols + col + 2] + new_image[(row*cols + col)]) / 4;

	__syncthreads();

}

__global__ void bilinear3(unsigned char *new_image, const unsigned char *image, int rows, int cols)
{
	int	col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;
//	int index = row*cols / 2 + col;
	row *= 2;
	col *= 2;
	unsigned char a, b;
	a = new_image[row*cols + col];
	b = new_image[(row + 1)*cols + col + 1];
	new_image[row*cols + col + 1] = (a + new_image[(row - 1)*cols + col + 1] + new_image[row*cols + col + 2] + b) / 4;

	new_image[(row + 1)*cols + col] = (a + new_image[(row + 1)*cols + col - 1] + b + new_image[(row + 2)*cols + col]) / 4;
	__syncthreads();

}
